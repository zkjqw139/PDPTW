#pragma  once
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "loadData.h"
#include "BaseInitOp.cuh"
#include "evaluate.cuh"
#include "RemoveOP.cuh"
#include "InsertOp.cuh"
#include "critetionOp.cuh"
#include "OptimalOp.cuh"
#include "loadDataFrame.cuh"
#include "stdlib.h"
#include <stdio.h>
#include <io.h>
#include <string>

#include "process.h"
#include "spyderMetroData.cuh"
#include "Companyfactory.cuh"
#include "curl/curl.h"

#include "request.h"
#include "Dispatch.h"
#include "dispatchTest.cuh"


#include "ALNS.cuh"
#include "Route.cuh"
#include "msgpack.hpp"
#include <sstream>
#include <fstream>
#include <Eigen/Dense>

#include "MCTBase.cuh"
#include "mctBaseTest.cuh"
#include <typeinfo>

//�ƽӲ�����
//�����ƽӲ����⣬��һ�����������ڳ�վ������D�ڸ�����˾λ�õ�,VRP����
//�����ƽӲ�������Ҫ�����Ŀ�꺯��
//�䳵�̶��ɱ�*�䳵��Ŀ+sum(ÿ�������ƶ�����*������̬�ɱ�)+��sum(���ͻ��ȴ�ʱ�����ָ������))
//Ŀ������С�������
//Լ������
//����Լ��
//1.��������ʱ��<30min
//2.3km <�������о���<20km
//3.40��<������������<60��
//4.��������һ���̶ȵĳ��ص��ǻ���ϳͷ�
//�켣Լ��
//5.����ÿһ����·����վ�����ٵ���һ��
//6.���г������Ǵӵ���վ�����ٻص�����վ

//��һȦ���ж���һȦ��Ӱ��
//���߹���
//ÿ12����ͳ�Ƶ���վ��������֪����վ���OD
//ʱ�� ״̬ OD�Ե�����
//��·����


//˫��滮ģ��
//����ֱ��
//�û��ȴ�ʱ�������
//�û��ȴ�ʱ�����

void showPipeFlow(std::vector<PDPTW::CompanyWithTimeTable> largeCompanys) {

	for (int i = 1; i < largeCompanys.size(); i++) {

		std::cout << largeCompanys[i].name << std::endl;
		for (int j = 0; j < largeCompanys[i].pipeflow.size(); j++) {

			std::cout << largeCompanys[i].pipeflow[j].waitDemand << "   ";

			
		}
		std::cout << std::endl;
	}


}


//����
PDPTW::Route  readRoute(std::string FileName,int rtype,int timeid) {


	ifstream ism(FileName);
	std::string line;
	int count = 0;

	//��·������Ϣ
	std::vector<int>  nodeIDs;                 //��·��������Ҫ�����վ��


	int  routeCount=0;                         //��ǰ��·�����Ĺ�˾��Ŀ
	int  routeDistance=0;                      //��·�����
	int  routeSingleDistance=0;

	int  routeDuration=0;                      //��·��ʻʱ��
	int  routeSingleDuration=0;

	int  routeType = rtype;                    //type 0:��Ƶ��Ƶ������
											   //type 1:��ʱ����


	std::vector<int>  stationWaitFlow;
	std::stringstream iss;

	while (std::getline(ism, line)) {
	    
		if(count==0)
			routeCount = stoi(line);

		 
		if (count == 1)
			routeDistance = stoi(line);
			
		if (count == 2)
			routeSingleDistance = stoi(line);
			
		if (count == 3)
			routeDuration = stoi(line);
			
		if (count == 4)
			routeSingleDuration = stoi(line);
			
		if (count == 5)
			routeType = stoi(line);
			
		if (count == 6) {
			std::stringstream iss(line);
			int number;
			while (iss >> number) {
				nodeIDs.push_back(number);
			}

		}
		if (count == 7){
			std::stringstream iss(line);
			int number;
			while (iss >> number) {
				stationWaitFlow.push_back(number);
			}
		}


		count = count + 1;
	}

	

	PDPTW::Route newRoute(nodeIDs, routeCount, routeDistance, routeSingleDistance, routeDuration, routeSingleDuration, routeType, stationWaitFlow, timeid);
	//newRoute.readShow();
	return newRoute;

}

//�������������ļ���
//����Ѱ����Ҫʹ�õ��ļ�
std::vector<PDPTW::Route> travelFile(std::string FileName, std::string parentFileName){

	std::vector<PDPTW::Route> routePool;
	//���ڲ��ҵľ��
	long    handle;
	struct  _finddata_t fileinfo;
	//��һ�β���
	handle = _findfirst(FileName.c_str(), &fileinfo);
	


	if (handle == -1) {
		std::cout << "no file found" << std::endl;
		return routePool;
	}

	while (!_findnext(handle, &fileinfo)){
		std::string originFileName = parentFileName + fileinfo.name;
	    
		int rtype = 0;

		if (fileinfo.name[0] != 's') {
			rtype = 0;
		}
		else {
			rtype = 1;
		}


		int timeid = 0;
		if (rtype == 0) {
			timeid = (fileinfo.name[0]-'0');
		}
		else {
			timeid = -1;
		}


		PDPTW::Route newRoute = readRoute(originFileName, rtype, timeid);
		routePool.push_back(newRoute);
	} 

	_findclose(handle);
	return routePool;
}


int main()
{   
	//���빫˾
	std::string  filename = "C:/Users/hasee/Documents/Visual Studio 2015/Projects/PDPTW/DataSrc/���˵���վ�Ӳ���վ��ͳ��.csv";
	vector<PDPTW::company> companys=PDPTW::loadDF(filename);
	
	

	std::string metro_station_name = "'����վ'";
	std::string hour_begin = "8";
	std::string hour_end = "9";
	std::string day_begin = "20190301";
	std::string day_end = "20190401";
	
	
	
	map<int, float> flowPercent = PDPTW::metroFlowDistribution();
	std::vector<PDPTW::station> stationPool;

	//���ڲ�ͬ��˾�������ɲ�ͬ�ľ������ڼ��㹫˾����
	std::string origins = "120.220429,30.187295";
	std::string dest = "120.189549,30.190514";
	std::string name = "����";
	PDPTW::CompanyWithTimeTable depot(name, 120.220429, 30.187295, 0, 0, -1);

	std::vector<PDPTW::CompanyWithTimeTable> smallCompanys;
	std::vector<PDPTW::CompanyWithTimeTable> largeCompanys;
	smallCompanys.push_back(depot);
	largeCompanys.push_back(depot);


	std::vector<PDPTW::CompanyWithTimeTable> companyWithTimeWindows;
	for (int i = 0; i < companys.size(); i++) {
		companyWithTimeWindows.push_back(PDPTW::companyFactory(companys[i], flowPercent));
	}
	
	//����ID��վ�����ƵĶ�Ӧ��ϵ
	std::map<int, std::string> smallCompanyID2Name;
	std::map<int, std::string> largeCompanyID2Name;
	std::map<std::string, int>  companysName2ID;

	for (int i = 0; i < companyWithTimeWindows.size(); i++) {
		if (companyWithTimeWindows[i].companyType == 0) {
			smallCompanys.push_back(companyWithTimeWindows[i]);

		}
		else {
			largeCompanys.push_back(companyWithTimeWindows[i]);
		}
	}

	//�ֵ�����
	for (int i = 0; i < smallCompanys.size(); i++) {
		std::string name = smallCompanys[i].name;
		int  index = i;
		smallCompanyID2Name.insert(pair<int, std::string>(i, name));
	}
	
	for (int i = 0; i < largeCompanys.size(); i++) {
		std::string name = largeCompanys[i].name;
		int index = i;
		largeCompanyID2Name.insert(pair<int, std::string>(i, name));
	}

	for (int i = 0; i < companys.size(); i++) {
		std::string name = companys[i].name;
		int index = i;
		companysName2ID.insert(pair<std::string, int>(name, i));
	}

	//�����ļ���
	std::string fileName = "route\\*";
	std::string parentFileName = "route\\";
	std::vector<PDPTW::Route> routePool = travelFile(fileName, parentFileName);


	for (int i = 0; i < companys.size(); i++) {
		std::deque<int>  flow = PDPTW::stationFlowGenerate(companys[i], flowPercent);
		std::string  name = companys[i].name;
		companys[i].employeeWantUseBusNum = companys[i].employeeNum / 15;
		int companyType = PDPTW::companyTypeDecision(companys[i]);
    
		PDPTW::station oneStation(name, flow, companyType);
		stationPool.push_back(oneStation);
		if (companyType == 2) {
			std::string destination = to_string(companys[i].lon) + "," + to_string(companys[i].lat);
			PDPTW::disInfo dinfo =PDPTW::getAllDistanceAndDuration(origins, destination);

			int  dist = dinfo.circleDistance;
			int  singleDist = dinfo.metroToDestDistance;

			int  duration = dinfo.circleDuration;
			int  singleDuration = dinfo.metroToDestDuration;

			int  routeCount = 1;

			std::vector<int> stationflow = PDPTW::createLargeCompanyRoute(companys[i], flowPercent);
			std::vector<int> nodIDS;

			nodIDS.push_back(i);


			PDPTW::Route largeCompanyRoute(nodIDS, 1, dist, singleDist, duration, singleDuration, 2, stationflow, -2);
			routePool.push_back(largeCompanyRoute);
		}
	}

	//IDת��
	for (int i = 0; i < routePool.size(); i++) {

		std::vector<int> routeIDs = routePool[i].nodeIDs;

		int routeType = routePool[i].routeType;
		if (routeType == 0) {

			for (int j = 0; j < routeIDs.size(); j++) {

				int rid = routeIDs[j];
				std::string newName = largeCompanyID2Name.at(rid);
				int newid = companysName2ID.at(newName);
				routeIDs[j] = newid;
			}			
			routePool[i].nodeIDs = routeIDs;
		}

		else if (routeType == 1) {

			for (int j = 0; j < routeIDs.size(); j++) {

				int rid = routeIDs[j];
				std::string newName = smallCompanyID2Name.at(rid);
				int newid = companysName2ID.at(newName);
				routeIDs[j] = newid;
			}
			routePool[i].nodeIDs = routeIDs;
		}
	}

	//���ɿ�������

	int segLength  = routePool.size(); //��·��Ŀ
	int timeLength = 30;               //����ʱ����Ŀ

	//��������
	Eigen::MatrixXd boardTable=Eigen::MatrixXd::Zero(segLength, timeLength);
	for (int i = 0; i < routePool.size(); i++) {	
		vector<int> stationWaitFlow = routePool[i].stationWaitFlow;
		for (int j = 0; j < stationWaitFlow.size(); j++) {
			boardTable(i, j) = stationWaitFlow[j];
		}
	}

	//վ���������
	std::vector<int> stationWaitFlow;
	for (int i = 0; i < companys.size(); i++) {
		stationWaitFlow.push_back(companys[i].employeeWantUseBusNum);
	}

	for (int i = 0; i < routePool.size(); i++) {
		routePool[i].setComapnyPercent(companys);
	} 
	
	//��˾��·ӳ���ֵ�����
	std::map<int, std::vector<int>> companytypeRouteDict; // ��˾����·��ID�ֵ�
	std::vector<int>  smallcompanyRouteID;                // С��˾��·ID
	std::vector<int>  largecompanyRouteID;                // ��˾��·ID
	std::vector<int>  middlecompanyRouteID;               // �й�˾��·ID

	for (int i = 0; i < routePool.size(); i++) {
		if (routePool[i].routeType == 0) {
			middlecompanyRouteID.push_back(i);
		}
		else if (routePool[i].routeType == 1) {
			smallcompanyRouteID.push_back(i);
		}
		else if (routePool[i].routeType == 2) {
			largecompanyRouteID.push_back(i);
		}
	}
	 
	companytypeRouteDict.insert(std::pair<int, std::vector<int>>(0, middlecompanyRouteID));
	companytypeRouteDict.insert(std::pair<int, std::vector<int>>(2, largecompanyRouteID));
	companytypeRouteDict.insert(std::pair<int, std::vector<int>>(1, smallcompanyRouteID));
	MCT::Board  newBoard(boardTable, stationWaitFlow);
	MCT::Agent  agent;
	MCT::BusPot pot;

	//̰������
	agent.BaseGreedySearch(newBoard, pot, companytypeRouteDict, routePool, true);
    

	system("pause");
    return 0;
}

 